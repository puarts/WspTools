

#include <wsp/gpu/WspFnGpuMatrix.h>
#include <wsp/mathutil/fn-math-gpl.h>



/*
    Transpose a 2D array (see SDK transpose example)
*/
void WspFnGpu::transpose(u32 *d_src, u32 *d_dest, u32 width, int height)
{
    dim3 grid(wsp::math::iDivUp(width, BLOCK_DIM), wsp::math::iDivUp(height, BLOCK_DIM), 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
    WspFnGpu::d_transpose<<< grid, threads >>>(d_dest, d_src, width, height);
    cutilCheckMsg("Kernel execution failed");
}