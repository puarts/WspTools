#include "hip/hip_runtime.h"
//! WspFnGpuImgFilter.cu
// @author Junichi Nishikata


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cutil_math.h>

#include <wsp/gpu/WspFnGpuImgFilter.h>
#include <wsp/gpu/WspFnGpuMatrix.h>
#include <wsp/mathutil/fn-math-gpl.h>

#define USE_SIMPLE_FILTER 1




/*
Perform Gaussian filter on a 2D image using CUDA

Parameters:
d_src  - pointer to input image in device memory
d_dest - pointer to destination image in device memory
d_temp - pointer to temporary storage in device memory
width  - image width
height - image height
sigma  - sigma of Gaussian
order  - filter order (0, 1 or 2)
*/

// 8-bit RGBA version
wsp::State WspFnGpu::GaussianFilterRGBA(u32 *h_dest, u32 *h_src, int width, int height, float sigma, int order, int nthreads)
{
    if(width<=0 || height<=0){ return WspStd::Failure; }

    u32* d_src, *d_dest, *d_temp;
    u32 length = width*height;

    size_t size = length * sizeof(float);

    // Allocate vectors in device memory
    cutilSafeCall( hipMalloc((void**)&d_src, size) );
    cutilSafeCall( hipMalloc((void**)&d_dest, size) );
    cutilSafeCall( hipMalloc((void**)&d_temp, size) );

    // Copy vectors from host memory to device memory
    cutilSafeCall( hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice) );

    WspFnGpu::GaussianFilterRGBA(d_src, d_dest, d_temp, width, height, sigma, order, nthreads);

    cutilSafeCall( hipMemcpy(h_dest, d_dest, size, hipMemcpyDeviceToHost) );

    cutilSafeCall(hipFree(d_src));
    cutilSafeCall(hipFree(d_dest));
    cutilSafeCall(hipFree(d_temp));

    cutilDeviceReset();

    return WSP_STATE_SUCCESS;
}

void WspFnGpu::GaussianFilterRGBA(u32 *d_src, u32 *d_dest, u32 *d_temp, int width, int height, float sigma, int order, int nthreads)
{
    // compute filter coefficients
    const float
        nsigma = sigma < 0.1f ? 0.1f : sigma,
        alpha = 1.695f / nsigma,
        ema = (float)exp(-alpha),
        ema2 = (float)exp(-2*alpha),
        b1 = -2*ema,
        b2 = ema2;

    float a0 = 0, a1 = 0, a2 = 0, a3 = 0, coefp = 0, coefn = 0;
    switch (order) {
        case 0: {
            const float k = (1-ema)*(1-ema)/(1+2*alpha*ema-ema2);
            a0 = k;
            a1 = k*(alpha-1)*ema;
            a2 = k*(alpha+1)*ema;
            a3 = -k*ema2;
            break;
        }

        case 1: {
            const float k = (1-ema)*(1-ema)/ema;
            a0 = k*ema;
            a1 = a3 = 0;
            a2 = -a0;
            break;
        }

        case 2: {
            const float
                ea = (float)exp(-alpha),
                k = -(ema2-1)/(2*alpha*ema),
                kn = (-2*(-1+3*ea-3*ea*ea+ea*ea*ea)/(3*ea+1+3*ea*ea+ea*ea*ea));
            a0 = kn;
            a1 = -kn*(1+k*alpha)*ema;
            a2 = kn*(1-k*alpha)*ema;
            a3 = -kn*ema2;
            break;
        }

        default:
            WSP_COMMON_ERROR_LOG("GaussianFilter: invalid order parameter!\n");
            return;
    }
    coefp = (a0+a1)/(1+b1+b2);
    coefn = (a2+a3)/(1+b1+b2);

    // process columns
#if USE_SIMPLE_FILTER
    WspFnGpu::d_simpleRecursive_rgba<<< wsp::math::iDivUp(width, nthreads), nthreads >>>(d_src, d_temp, width, height, ema);
#else
    WspFnGpu::d_recursiveGaussian_rgba<<< wsp::math::iDivUp(width, nthreads), nthreads >>>(d_src, d_temp, width, height, a0, a1, a2, a3, b1, b2, coefp, coefn);
#endif
    cutilCheckMsg("Kernel execution failed");

    WspFnGpu::transpose(d_temp, d_dest, width, height);
    cutilCheckMsg("transpose: Kernel execution failed");

    // process rows
#if USE_SIMPLE_FILTER
    WspFnGpu::d_simpleRecursive_rgba<<< wsp::math::iDivUp(height, nthreads), nthreads >>>(d_dest, d_temp, height, width, ema);
#else
    WspFnGpu::d_recursiveGaussian_rgba<<< wsp::math::iDivUp(height, nthreads), nthreads >>>(d_dest, d_temp, height, width, a0, a1, a2, a3, b1, b2, coefp, coefn);
#endif
    cutilCheckMsg("Kernel execution failed");

    WspFnGpu::transpose(d_temp, d_dest, height, width);
}