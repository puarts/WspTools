#include "hip/hip_runtime.h"
//! WspFnGpuImgFilter_kernel.cu
// @author Junichi Nishikata

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cutil_math.h>

#include <wsp/gpu/WspFnGpuImgFilter.h>
#include "WspFnGpuInline.cu"

/*
    simple 1st order recursive filter
    - processes one image column per thread

    parameters:    
    id - pointer to input data (RGBA image packed into 32-bit integers)
    od - pointer to output data 
    w  - image width
    h  - image height
    a  - blur parameter
*/
__global__ void WspFnGpu::d_simpleRecursive_rgba(u32 *d_src, u32 *d_dst, int w, int h, float a)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    if (x >= w){ return; }
    
    d_src += x;    // advance pointers to correct column
    d_dst += x;

    // forward pass
    float4 pix_prev = RgbaIntToFloat(*d_src);  // previous output
    for (int y = 0; y < h; y++) {
        float4 pix = RgbaIntToFloat(*d_src);
        float4 pix_dst = pix + a*(pix_prev - pix);   // simple lerp between current and previous value
        *d_dst = RgbaFloatToInt(pix_dst);
        d_src += w; d_dst += w;    // move to next row
        pix_prev = pix_dst;
    }

    // reset pointers to point to last element in column
    d_src -= w;
    d_dst -= w;

    // reverse pass
    // ensures response is symmetrical
    pix_prev = RgbaIntToFloat(*d_src);
    for (int y = h-1; y >= 0; y--) {
        float4 pix = RgbaIntToFloat(*d_src);
        float4 pix_new = pix + a*(pix_prev - pix);
        *d_dst = RgbaFloatToInt((RgbaIntToFloat(*d_dst) + pix_new)*0.5f);
        d_src -= w; d_dst -= w;  // move to previous row
        pix_prev = pix_new;
    }
}

/*
    recursive Gaussian filter

    parameters:    
    id - pointer to input data (RGBA image packed into 32-bit integers)
    od - pointer to output data 
    w  - image width
    h  - image height
    a0-a3, b1, b2, coefp, coefn - filter parameters
*/

__global__ void
WspFnGpu::d_recursiveGaussian_rgba(u32 *id, u32 *od, int w, int h, float a0, float a1, float a2, float a3, float b1, float b2, float coefp, float coefn)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    if (x >= w) return;
    
    id += x;    // advance pointers to correct column
    od += x;

    // forward pass
    float4 xp = make_float4(0.0f);  // previous input
    float4 yp = make_float4(0.0f);  // previous output
    float4 yb = make_float4(0.0f);  // previous output by 2
#if CLAMP_TO_EDGE
    xp = RgbaIntToFloat(*id); yb = coefp*xp; yp = yb;
#endif
    for (int y = 0; y < h; y++) {
        float4 xc = RgbaIntToFloat(*id);
        float4 yc = a0*xc + a1*xp - b1*yp - b2*yb;
        *od = RgbaFloatToInt(yc);
        id += w; od += w;    // move to next row
        xp = xc; yb = yp; yp = yc; 
    }

    // reset pointers to point to last element in column
    id -= w;
    od -= w;

    // reverse pass
    // ensures response is symmetrical
    float4 xn = make_float4(0.0f);
    float4 xa = make_float4(0.0f);
    float4 yn = make_float4(0.0f);
    float4 ya = make_float4(0.0f);
#if CLAMP_TO_EDGE
    xn = xa = RgbaIntToFloat(*id); yn = coefn*xn; ya = yn;
#endif
    for (int y = h-1; y >= 0; y--) {
        float4 xc = RgbaIntToFloat(*id);
        float4 yc = a2*xn + a3*xa - b1*yn - b2*ya;
        xa = xn; xn = xc; ya = yn; yn = yc;
        *od = RgbaFloatToInt(RgbaIntToFloat(*od) + yc);
        id -= w; od -= w;  // move to previous row
    }
}
