#include "hip/hip_runtime.h"
//! WspFnGpuVector_kernel.cu
// @author Junichi Nishikata

#ifndef _WspFnGpuVector_kernel_CU_
#define _WspFnGpuVector_kernel_CU_

#include <wsp/gpu/WspFnGpuVector.h>

// Device code
__global__ void WspFnGpu::VectorAdd_d(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N){
        C[i] = A[i] + B[i];
    }
}

#endif