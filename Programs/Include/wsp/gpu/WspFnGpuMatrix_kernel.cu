#include "hip/hip_runtime.h"
//! WspFnGpuMatrix_kernel.cu
// @author Junichi Nishikata


#include <wsp/gpu/WspFnGpuMatrix.h>


// Transpose kernel (see transpose SDK sample for details)
__global__ void WspFnGpu::d_transpose(u32 *odata, u32 *idata, int width, int height)
{
    __shared__ u32 block[BLOCK_DIM][BLOCK_DIM+1];
    
    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
    if((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}