
#include <stdio.h>
#include <cutil_inline.h>

#include <wsp/gpu/WspFnGpuVector.h>


wsp::State wsp::gpu::VectorAdd(float *o_vec, float *in_vecA, float *in_vecB, int length){
    float* d_A, *d_B, *d_C;

    size_t size = length * sizeof(float);

    // Allocate vectors in device memory
    cutilSafeCall( hipMalloc((void**)&d_A, size) );
    cutilSafeCall( hipMalloc((void**)&d_B, size) );
    cutilSafeCall( hipMalloc((void**)&d_C, size) );

    // Copy vectors from host memory to device memory
    cutilSafeCall( hipMemcpy(d_A, in_vecA, size, hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(d_B, in_vecB, size, hipMemcpyHostToDevice) );

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;
    wsp::gpu::VectorAdd_d<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, length);
    cutilCheckMsg("kernel launch failure");

#ifdef _DEBUG
    cutilSafeCall( cutilDeviceSynchronize() );
#endif

    // Copy result from device memory to host memory
    cutilSafeCall( hipMemcpy(o_vec, d_C, size, hipMemcpyDeviceToHost) );
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    cutilDeviceReset();

    return WSP_STATE_SUCCESS;
}
