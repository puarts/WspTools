#include "hip/hip_runtime.h"

#include <WspGpu/WspGpuDefine.h>



// RGBA version
// reads from 32-bit u32 array holding 8-bit RGBA

// convert floating point rgba color to 32-bit integer
inline __device__ u32 WspFnGpu::RgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (u32(rgba.w*255)<<24) | (u32(rgba.z*255)<<16) | (u32(rgba.y*255)<<8) | u32(rgba.x*255);
}

// convert from 32-bit int to float4
inline __device__ float4 WspFnGpu::RgbaIntToFloat(u32 c)
{
    float4 rgba;
    rgba.x = (c & 0xff) / 255.0f;
    rgba.y = ((c>>8) & 0xff) / 255.0f;
    rgba.z = ((c>>16) & 0xff) / 255.0f;
    rgba.w = ((c>>24) & 0xff) / 255.0f;
    return rgba;
}